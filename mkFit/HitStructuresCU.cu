
#include <vector>
#include <algorithm>

#include "HitStructuresCU.h"

void LayerOfHitsCU::alloc_hits(int size) {
  hipMalloc((void**)&m_hits, sizeof(Hit)*size);
  m_capacity = size;
}

void LayerOfHitsCU::free_hits() {
  hipFree(m_hits);
  m_capacity = 0;
}

void LayerOfHitsCU::alloc_phi_bin_infos(int nz, int nphi) {
  hipMalloc((void**)&m_phi_bin_infos, sizeof(PairIntsCU)*nz*nphi);
  m_nz = nz;
}

void LayerOfHitsCU::free_phi_bin_infos() {
  hipFree(m_phi_bin_infos);
  m_nz = 0;
}

void LayerOfHitsCU::copyLayerOfHitsFromCPU(LayerOfHits &layer) {
  hipMemcpy(m_hits, layer.m_hits, sizeof(Hit)*m_capacity, hipMemcpyHostToDevice);
  // TODO: probably quite inefficient:
  for (int i = 0; i < m_nz; ++i) {
    hipMemcpy(m_phi_bin_infos + i*m_nphi, &(layer.m_phi_bin_infos[i][0]), sizeof(int)*m_nphi, hipMemcpyHostToDevice);
  }
}

#if 0
BunchOfHitsCU::BunchOfHitsCU() :
      m_real_size {Config::maxHitsPerBunch}, m_fill_index {0} {
  hipMalloc((void**)&m_hits, sizeof(Hit)*m_real_size);
}

BunchOfHitsCU::~BunchOfHitsCU() {
  hipFree(m_hits);
  m_fill_index = 0;
}

void BunchOfHitsCU::copyBunchOfHitsFromCPU(BunchOfHits& bunch) {
  m_fill_index = bunch.m_fill_index;
  hipMemcpy(m_hits, bunch.m_hits, sizeof(Hit)*m_fill_index, hipMemcpyHostToDevice);
}

void BunchOfHitsCU::allocatePhiBinInfos(int num_phi_bins) {
  this->num_phi_bins = num_phi_bins;
  hipMalloc((void**)&m_phi_bin_infos_first, sizeof(int)*num_phi_bins);
  hipMalloc((void**)&m_phi_bin_infos_second, sizeof(int)*num_phi_bins);
}

void BunchOfHitsCU::freePhiBinInfos() {
  hipFree(m_phi_bin_infos_first);
  hipFree(m_phi_bin_infos_second);
}

void BunchOfHitsCU::copyPhiBinInfosFromCPU(BunchOfHits &bunch) {
  // Strip the bin_infos pairs into two separate vectors
  // We cannot use std::pair on the GPU
  std::vector<int> first(num_phi_bins);
  std::vector<int> second(num_phi_bins);

  for (int i = 0; i < num_phi_bins; ++i) {
    std::pair<int, int> &infos = bunch.m_phi_bin_infos[i];  
    first[i] = infos.first;
    second[i] = infos.second;
  }

  hipMemcpy(m_phi_bin_infos_first, &first[0], sizeof(int)*num_phi_bins, hipMemcpyHostToDevice);
  hipMemcpy(m_phi_bin_infos_second, &second[0], sizeof(int)*num_phi_bins, hipMemcpyHostToDevice);
}
#endif
