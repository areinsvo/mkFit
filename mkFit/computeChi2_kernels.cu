#include "hip/hip_runtime.h"
#include "computeChi2_kernels.h"

#include "GPlex.h"
#include "kalmanUpdater_kernels.h"
#include "gpu_utils.h"

#include <stdio.h>

#define L 6
#define HS 6
#define HV 3
#define BLOCK_SIZE_X 256


__device__ void chi2Similarity_fn(
    const GPlexReg2V &a,
    const GPlexReg2S &c, // in registers
    float *d, const size_t dN) {

  int n = threadIdx.x + blockIdx.x * blockDim.x;

  // manually subrtact into local vars -- 3 of them
  /*float x0 = a[0 * aN + n] - b[0 * aN + n];*/
  /*float x1 = a[1 * aN + n] - b[1 * aN + n];*/
  /*float x2 = a[2 * aN + n] - b[2 * aN + n];*/
  /*d[0 * dN + n] = c[0]*x0*x0 + c[2]*x1*x1 + c[5]*x2*x2 +*/
              /*2*( c[1]*x1*x0 + c[3]*x2*x0 + c[4]*x1*x2);*/
  d[0 * dN + n] = c[0]*a[0]*a[0]
                + c[2]*a[1]*a[1] 
            + 2*( c[1]*a[1]*a[0]);
}


__device__ void RotateResidulsOnTangentPlane_fn(const float r00,//r00
				  const float r01,//r01
				  const GPlexRegHV &a  ,//res_glo
          GPlexReg2V &b  )//res_loc
{

   // res_loc = rotT * res_glo
   //   B     =  R   *    A   
  b[0] =  r00*a[0] + r01*a[1];
  b[1] =  a[2];
}


__device__ void ProjectResErr_fn(const float a00,
		   const float a01,
		   const GPlexRegHS &b, 
       GPlexRegHH &c)
{
  // C = A * B, C is 3x3, A is 3x3 , B is 3x3 sym

  // Based on script generation and adapted to custom sizes.
      c[ 0] = a00*b[ 0] + a01*b[ 1];
      c[ 1] = a00*b[ 1] + a01*b[ 2];
      c[ 2] = a00*b[ 3] + a01*b[ 4];
      c[ 3] = b[ 3];
      c[ 4] = b[ 4];
      c[ 5] = b[ 5];
      c[ 6] = a01*b[ 0] - a00*b[ 1];
      c[ 7] = a01*b[ 1] - a00*b[ 2];
      c[ 8] = a01*b[ 3] - a00*b[ 4];
}


__device__ void ProjectResErrTransp_fn(const float a00,
			 const float a01, const GPlexRegHH &b, GPlexReg2S &c)
{
  // C = A * B, C is 3x3 sym, A is 3x3 , B is 3x3

  // Based on script generation and adapted to custom sizes.
      c[ 0] = b[ 0]*a00 + b[ 1]*a01;
      c[ 1] = b[ 3]*a00 + b[ 4]*a01;
      c[ 2] = b[ 5];
}


__device__ void computeChi2_fn(
    const GPlexLS &propErr, const GPlexHS &msErr, const GPlexHV &msPar,
    const GPlexLV &propPar, GPlexQF &outChi2, const int N) {
  int grid_width = blockDim.x * gridDim.x;
  int n = threadIdx.x + blockIdx.x * blockDim.x;
  /*float resErr_reg[HS]; // ~ resErr_glo*/
  GPlexRegHS resErr_reg;

  for (int z = 0; z < (N-1)/grid_width  +1; z++) {
    n += z*grid_width;

    if (n < N) {

      // coordinate change
      float rotT00;
      float rotT01;
      const float r = hipo(msPar(n, 0, 0), msPar(n, 1, 0));
      rotT00 = -(msPar(n, 1, 0) + propPar(n, 1, 0))/(2*r);
      rotT01 =  (msPar(n, 0, 0) + propPar(n, 0, 0))/(2*r);

      /*float res_glo[HV];*/
      GPlexRegHV res_glo;
      subtractFirst3_fn(msPar, propPar, res_glo, N, n);

      for (int j = 0; j < HS; ++j) {
        resErr_reg[j] = 0; //resErr[j*resErr_stride + n];
      }
      addIntoUpperLeft3x3_fn(propErr, msErr, resErr_reg, N, n);

      GPlexReg2V res_loc;   //position residual in local coordinates
      RotateResidulsOnTangentPlane_fn(rotT00,rotT01,res_glo,res_loc);
      /*MPlex2S resErr_loc;//covariance sum in local position coordinates*/
      /*MPlexHH tempHH;*/
      GPlexReg2S resErr_loc; // 2x2 sym
      GPlexRegHH tempHH;  // 3*3 sym
      ProjectResErr_fn  (rotT00, rotT01, resErr_reg, tempHH);
      ProjectResErrTransp_fn(rotT00, rotT01, tempHH, resErr_loc);

      /*invertCramerSym_fn(resErr_reg);*/
      invertCramerSym2x2_fn(resErr_loc);

      chi2Similarity_fn(res_loc, resErr_loc, outChi2.ptr, outChi2.stride);
    }
  }
}


__global__ void computeChi2_kernel(
    const GPlexLS propErr, const GPlexHS msErr, const GPlexHV msPar, 
    const GPlexLV propPar, GPlexQF outChi2, const int N) {
  int itrack = threadIdx.x + blockDim.x*blockIdx.x;
  if (itrack < N) {
    computeChi2_fn
      (propErr, msErr, msPar, propPar,
       outChi2, N);
  }
}


void computeChi2_wrapper(hipStream_t &stream, 
    const GPlexLS &propErr, const GPlexHS &msErr,
    const GPlexHV &msPar, const GPlexLV &propPar, GPlexQF &outChi2,
    const int N) {
  int gridx = std::min((N-1)/BLOCK_SIZE_X + 1,
                       max_blocks_x);
  dim3 grid(gridx, 1, 1);
  dim3 block(BLOCK_SIZE_X, 1, 1);
  computeChi2_kernel <<< grid, block, 0, stream >>>
    (propErr, msErr, msPar, propPar, outChi2, N);
 }
