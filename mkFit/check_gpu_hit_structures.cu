#include "hip/hip_runtime.h"
#include "check_gpu_hit_structures.h"

/*#include "reorganize_gplex.cu"*/
#include "HitStructures.h"
#include "HitStructuresCU.h"
#include "reorganize_gplex.h"

#include <iostream>


__global__ void get_hit_pos_and_err(LayerOfHitsCU *layers,
    int ilay, int hit_idx, float *pos, float *err, int pos_size, int err_size) {
  if (threadIdx.x + blockDim.x * blockIdx.x == 0) {
    LayerOfHitsCU &layer = layers[ilay];
    Hit &hit = layer.m_hits[hit_idx];
    float *posArray = get_posArray(hit);
    float *errArray = get_errArray(hit);
    for (int i = 0; i < pos_size; ++i) {
      pos[i] = posArray[i];
    }
    for (int i = 0; i < err_size; ++i) {
      err[i] = errArray[i];
    }
  }
}


void compare_carrays(const float *h_a, const float *d_a, 
                     const float prec, const int n) 
{
  for (int i = 0; i < n; ++i) {
    // should be relative comparison, verify if div by 0 will happen
    if (std::abs(h_a[i] - d_a[i]) > prec) {
      std::cerr << i << " : " << h_a[i] << " / " << d_a[i] << std::endl;
    }
  }
}


void check_event_of_hits_gpu(const EventOfHits& event_of_hits)
{
  EventOfHitsCU event_of_hits_cu;
  event_of_hits_cu.allocGPU(event_of_hits);
  event_of_hits_cu.copyFromCPU(event_of_hits);

  constexpr int pos_size = 3;
  constexpr int err_size = 6;

  float *d_pos, *d_err;
  float pos[pos_size], err[err_size];

  hipMalloc((void**)&d_pos, pos_size*sizeof(float));
  hipMalloc((void**)&d_err, err_size*sizeof(float));

  dim3 grid(1, 1, 1);
  dim3 block(1, 1, 1);

  int ilay = 2;
  int hit_idx = 3;

  get_hit_pos_and_err <<< grid, block >>>
    (event_of_hits_cu.m_layers_of_hits, ilay, hit_idx, d_pos, d_err, pos_size, err_size);

  hipMemcpy(pos, d_pos, pos_size*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(err, d_err, err_size*sizeof(float), hipMemcpyDeviceToHost);

  compare_carrays(event_of_hits.m_layers_of_hits[ilay].m_hits[hit_idx].posArray(),
                  pos, 1e-3, pos_size);
  compare_carrays(event_of_hits.m_layers_of_hits[ilay].m_hits[hit_idx].errArray(),
                  err, 1e-3, err_size);

  hipFree(d_pos);
  hipFree(d_err);

  event_of_hits_cu.deallocGPU();
}
