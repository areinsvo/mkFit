#include "hip/hip_runtime.h"
#include "reorganize_gplex.h"
#include <stdio.h>

#include "FitterCU.h"
#include "accessors_cu.h"
#include "Track.h"
#include "gpu_utils.h"

__device__ float *get_posArray(Hit &hit) {
    return hit.posArrayCU();
}
__device__ float *get_errArray(Hit &hit) {
    return hit.errArrayCU();
}

__device__ float *get_posArray(Track &track) {
    return track.posArrayCU();
}
__device__ float *get_errArray(Track &track) {
    return track.errArrayCU();
}

template <typename GPlexObj>
__device__ void SlurpIn_fn(GPlexObj to, // float *fArray, int stride, int kSize, 
                           const char *arr, const int *vi, const int N) {
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  if (j<N) {
    const int *XHitPos = vi;
    const int off = XHitPos[j] * sizeof(Hit);
    for (int i = 0; i < to.kSize; ++i) { // plex_size
      to[j + to.stride*i] = * (decltype(to.ptr)) (arr + i*sizeof(decltype(*to.ptr)) + off);
    }
  }
}


template <typename GPlexObj>
__device__ void SlurpInIdx_fn(GPlexObj to,
                             const char *arr, const int idx, const int N) {
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  if (j<N) {
    for (int i = 0; i < to.kSize; ++i) { // plex_size
      auto tmp = * (decltype(to.ptr)) (arr + i*sizeof(decltype(*to.ptr)) + idx);
      to[j + to.stride*i] = tmp;
    }
  }
}


template <typename GPlexObj>
__device__ void SlurpOutIdx_fn(GPlexObj from, // float *fArray, int stride, int kSize, 
                               const char *arr, const int idx, const int N) {
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  if (j<N) {
    for (int i = 0; i < from.kSize; ++i) { // plex_size
      * (decltype(from.ptr)) (arr + i*sizeof(decltype(*from.ptr)) + idx) = from[j + from.stride*i];
    }
  }
}


__device__
void GetHitErr(GPlexHS& msErr, const char* array, const int beg, const int end)
{
      SlurpInIdx_fn(msErr, array, beg, end);
}


__device__
void GetHitPar(GPlexHV& msPar, const char* array, const int beg, const int end)
{
      SlurpInIdx_fn(msPar, array, beg, end);
}

__device__ void HitToMs_fn(GPlexHS &msErr, GPlexHV &msPar,
                           Hit *hits, const GPlexQI &XHitSize, 
                           const GPlexHitIdx &XHitArr, 
                           GPlexQI &HitsIdx, const int hit_cnt, 
                           const int itrack, const int N) {
  if (itrack < N) {

    const char *varr      = (char*) hits;
    const int   off_error = (char*) hits[0].errArrayCU() - varr;
    const int   off_param = (char*) hits[0].posArrayCU() - varr;

    if (hit_cnt < XHitSize[itrack]) {
      HitsIdx[itrack] = XHitArr(itrack, hit_cnt, 0) * sizeof(Hit);
    }
    SlurpInIdx_fn(msErr, varr + off_error, HitsIdx[itrack], N);
    SlurpInIdx_fn(msPar, varr + off_param, HitsIdx[itrack], N);
  }
}


__global__ void HitToMs_kernel(GPlexHS msErr, GPlexHV msPar, Hit *hits,
                               const GPlexQI XHitSize, const GPlexHitIdx XHitArr,
                               GPlexQI HitsIdx, const int hit_cnt, const int N) {
  int itrack = threadIdx.x + blockDim.x * blockIdx.x;
  HitToMs_fn(msErr, msPar, hits, XHitSize, XHitArr, HitsIdx, hit_cnt, itrack, N);
}


void HitToMs_wrapper(const hipStream_t& stream,
                     GPlexHS &msErr, GPlexHV &msPar, LayerOfHitsCU &layer, 
                     const GPlexQI &XHitSize, const GPlexHitIdx &XHitArr,
                     GPlexQI &HitsIdx, int hit_cnt, const int N) {
  int gridx = std::min((N-1)/BLOCK_SIZE_X + 1,
                       max_blocks_x);
  dim3 grid(gridx, 1, 1);
  dim3 block(BLOCK_SIZE_X, 1, 1);
  HitToMs_kernel <<< grid, block, 0 , stream >>>
    (msErr, msPar, layer.m_hits.data(), XHitSize, XHitArr, HitsIdx, hit_cnt, N);
  /*hipDeviceSynchronize();*/
}


__device__ void InputTracksCU_fn (Track *tracks, 
                                  GPlexLS &Err_iP, GPlexLV &Par_iP,
                                  GPlexQI &Chg, GPlexQF &Chi2,
                                  GPlexQI &Label, GPlexQI *HitsIdx,
                                  const int beg, const int end, 
                                  const int itrack, const int N) {
  if (itrack < (end-beg) && itrack < N) {
    Track &trk = tracks[beg];
    const char *varr       = (char*) &trk;
    int   off_error = (char*) trk.errArrayCU() - varr;
    int   off_param = (char*) trk.posArrayCU() - varr;

    int i= itrack + beg;
    const Track &trk_i = tracks[i];
    int idx = (char*) &trk_i - varr;

    Label(itrack, 0, 0) = tracks[i].label();
    Chg(itrack, 0, 0) = tracks[i].charge();
    Chi2(itrack, 0, 0) = tracks[i].chi2();
    SlurpInIdx_fn(Err_iP, varr + off_error, idx, N);
    SlurpInIdx_fn(Par_iP, varr + off_param, idx, N);

    for (int hi = 0; hi < 3; ++hi)
      HitsIdx[hi](itrack, 0, 0) = tracks[i].getHitIdx(hi);//dummy value for now
  }
}


__global__ void InputTracksCU_kernel(Track *tracks, 
                                     GPlexLS Err_iP, GPlexLV Par_iP,
                                     GPlexQI Chg, GPlexQF Chi2, GPlexQI Label,
                                     GPlexQI *HitsIdx,
                                     int beg, int end, int N) {
  int itrack = threadIdx.x + blockDim.x*blockIdx.x;
  InputTracksCU_fn(tracks, Err_iP, Par_iP, Chg, Chi2, Label, HitsIdx, beg, end, itrack, N);
}


void InputTracksCU_wrapper(const hipStream_t &stream, 
                           const EtaBinOfCandidatesCU &etaBin,
                           GPlexLS &Err_iP, GPlexLV &Par_iP,
                           GPlexQI &Chg, GPlexQF &Chi2, GPlexQI &Label,
                           GPlexQI *HitsIdx,
                           const int beg, const int end, const bool inputProp, int N) {
  int gridx = std::min((N-1)/BLOCK_SIZE_X + 1,
                       max_blocks_x);
  dim3 grid(gridx, 1, 1);
  dim3 block(BLOCK_SIZE_X, 1, 1);

  InputTracksCU_kernel <<< grid, block, 0, stream >>>
    (etaBin.m_candidates, Err_iP, Par_iP, Chg, Chi2, Label, HitsIdx,
     beg, end, N);
}


__device__ void InputTracksAndHitsCU_fn (Track *tracks, LayerOfHitsCU *layerHits,
                                         GPlexLS &Err_iP, GPlexLV &Par_iP,
                                         GPlexHS *msErr_arr, GPlexHV *msPar_arr,
                                         GPlexQI &Chg, GPlexQF &Chi2,
                                         GPlexQI &Label, GPlexQI *HitsIdx,
                                         const int beg, const int end, 
                                         const int itrack, const int N) {
  if (itrack < (end-beg) && itrack < N) {
    Track &trk = tracks[beg];
    const char *varr       = (char*) &trk;
    int   off_error = (char*) trk.errArrayCU() - varr;
    int   off_param = (char*) trk.posArrayCU() - varr;

    int i= itrack + beg;
    const Track &trk_i = tracks[i];
    int idx = (char*) &trk_i - varr;

    Label(itrack, 0, 0) = tracks[i].label();
    Chg(itrack, 0, 0) = tracks[i].charge();
    Chi2(itrack, 0, 0) = tracks[i].chi2();
    SlurpInIdx_fn(Err_iP, varr + off_error, idx, N);
    SlurpInIdx_fn(Par_iP, varr + off_param, idx, N);

    // Note Config::nLayers -- not suitable for building
    for (int hi = 0; hi < Config::nLayers; ++hi) {
      int hidx = tracks[i].getHitIdx(hi);
      Hit &hit = layerHits[hi].m_hits[hidx];

      HitsIdx[hi](itrack, 0, 0) = idx;
      if (hidx < 0) continue;

      SlurpInIdx_fn(msErr_arr[hi], (char *)hit.errArrayCU(), 0, N);
      SlurpInIdx_fn(msPar_arr[hi], (char *)hit.posArrayCU(), 0, N);
    }
  }
}


__global__ void InputTracksAndHitsCU_kernel(Track *tracks, LayerOfHitsCU *layers,
                                            GPlexLS Err_iP, GPlexLV Par_iP,
                                            GPlexHS *msErr_arr, GPlexHV *msPar_arr,
                                            GPlexQI Chg, GPlexQF Chi2, GPlexQI Label,
                                            GPlexQI *HitsIdx,
                                            int beg, int end, int N) {
  int itrack = threadIdx.x + blockDim.x*blockIdx.x;
  InputTracksAndHitsCU_fn(tracks, layers, Err_iP, Par_iP, msErr_arr, msPar_arr,
                          Chg, Chi2, Label, HitsIdx, beg, end, itrack, N);
}


void InputTracksAndHitsCU_wrapper(const hipStream_t &stream, 
                                  Track *tracks, EventOfHitsCU &event_of_hits,
                                  GPlexLS &Err_iP, GPlexLV &Par_iP,
                                  GPlexHS *msErr_arr, GPlexHV *msPar_arr,
                                  GPlexQI &Chg, GPlexQF &Chi2, GPlexQI &Label,
                                  GPlexQI *HitsIdx,
                                  const int beg, const int end, 
                                  const bool inputProp, int N) {
  int gridx = std::min((N-1)/BLOCK_SIZE_X + 1,
                       max_blocks_x);
  dim3 grid(gridx, 1, 1);
  dim3 block(BLOCK_SIZE_X, 1, 1);

  InputTracksAndHitsCU_kernel <<< grid, block, 0, stream >>>
    (tracks, event_of_hits.m_layers_of_hits.data(),
     Err_iP, Par_iP, 
     msErr_arr, msPar_arr, 
     Chg, Chi2, Label, HitsIdx,
     beg, end, N);
}


__device__ void OutputParErrCU_fn(Track *tracks, 
                                  const GPlexLS &Err, const GPlexLV &Par,
                                  const int beg, const int end, 
                                  const int itrack_plex, const int N) {
  Track &trk = tracks[beg];
  const char *varr       = (char*) &trk;
  int   off_error = (char*) trk.errArrayCU() - varr;
  int   off_param = (char*) trk.posArrayCU() - varr;

  int i= itrack_plex + beg;
  const Track &trk_i = tracks[i];
  int idx = (char*) &trk_i - varr;

  SlurpOutIdx_fn(Err, varr + off_error, idx, N);
  SlurpOutIdx_fn(Par, varr + off_param, idx, N);
}


__device__ void OutputTracksCU_fn(Track *tracks, 
                                  const GPlexLS &Err_iP, const GPlexLV &Par_iP,
                                  const GPlexQI &Chg, const GPlexQF &Chi2,
                                  const GPlexQI &Label, const GPlexQI *HitsIdx,
                                  const int beg, const int end, 
                                  const int itrack, const int N,
                                  const bool update_hit_idx) {
  if (itrack < (end-beg) && itrack < N) {
    Track &trk = tracks[beg];
    const char *varr       = (char*) &trk;
    int   off_error = (char*) trk.errArrayCU() - varr;
    int   off_param = (char*) trk.posArrayCU() - varr;

    int i= itrack + beg;
    const Track &trk_i = tracks[i];
    int idx = (char*) &trk_i - varr;

    SlurpOutIdx_fn(Err_iP, varr + off_error, idx, N);
    SlurpOutIdx_fn(Par_iP, varr + off_param, idx, N);
    tracks[i].setCharge(Chg(itrack, 0, 0));
    tracks[i].setChi2(Chi2(itrack, 0, 0));
    tracks[i].setLabel(Label(itrack, 0, 0));

    if (update_hit_idx) {
      tracks[i].resetHits();
      /*int nGoodItIdx = 0;*/
      for (int hi = 0; hi < Config::nLayers; ++hi) {
        tracks[i].addHitIdx(HitsIdx[hi](itrack, 0, 0),0.);
        // FIXME: We probably want to use registers instead of going for class members:
        /*int hit_idx = HitsIdx[hi](itrack, 0, 0);*/
        /*tracks[i].setHitIdx(hi, hit_idx);*/
        /*if (hit_idx >= 0) {*/
        /*nGoodItIdx++; */
        /*}*/
      }
      /*tracks[i].setNGoodHitIdx(nGoodItIdx);*/
      /*tracks[i].setChi2(0.);*/
    }
  }
}

__global__ void OutputTracksCU_kernel(Track *tracks, 
                                     GPlexLS Err_iP, GPlexLV Par_iP,
                                     GPlexQI Chg, GPlexQF Chi2, GPlexQI Label,
                                     GPlexQI *HitsIdx,
                                     int beg, int end, int N,
                                     const bool update_hit_idx=true) {
  int itrack = threadIdx.x + blockDim.x*blockIdx.x;
  OutputTracksCU_fn(tracks, Err_iP, Par_iP, Chg, Chi2, Label, HitsIdx,
                    beg, end, itrack, N, update_hit_idx);
}


void OutputTracksCU_wrapper(const hipStream_t &stream,
                            EtaBinOfCandidatesCU &etaBin,
                            GPlexLS &Err_iP, GPlexLV &Par_iP,
                            GPlexQI &Chg, GPlexQF &Chi2, GPlexQI &Label,
                            GPlexQI *HitsIdx,
                            const int beg, const int end, const bool outputProp, int N) {
  int gridx = std::min((N-1)/BLOCK_SIZE_X + 1,
                       max_blocks_x);
  dim3 grid(gridx, 1, 1);
  dim3 block(BLOCK_SIZE_X, 1, 1);

  OutputTracksCU_kernel <<< grid, block, 0, stream >>>
    (etaBin.m_candidates, Err_iP, Par_iP, Chg, Chi2, Label, HitsIdx, beg, end, N);
}


void OutputFittedTracksCU_wrapper(const hipStream_t &stream,
                                  Track *tracks_cu, 
                                  GPlexLS &Err_iP, GPlexLV &Par_iP,
                                  GPlexQI &Chg, GPlexQF &Chi2, GPlexQI &Label,
                                  const int beg, const int end, int N) {
  int gridx = std::min((N-1)/BLOCK_SIZE_X + 1,
                       max_blocks_x);
  dim3 grid(gridx, 1, 1);
  dim3 block(BLOCK_SIZE_X, 1, 1);

  OutputTracksCU_kernel <<< grid, block, 0, stream >>>
    (tracks_cu, Err_iP, Par_iP, Chg, Chi2, Label, nullptr, beg, end, N, false);
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

// m_tracks_per_seed: play the same role than seed_cand_idx in the cpu code
__device__ void InputTracksAndHitIdxComb_fn(Track *tracks, int *m_tracks_per_seed,
                                            GPlexLS &Err_iP, GPlexLV &Par_iP,
                                            GPlexQI &Chg, GPlexQF &Chi2,
                                            GPlexQI &Label, GPlexQI *HitsIdx,
                                            GPlexQI &SeedIdx, GPlexQI &CandIdx,
                                            GPlexQB &Valid,
                                            const int Nhits,
                                            const int beg, const int end, 
                                            const int itrack_plex, const int N)
{
  if (itrack_plex < N) {
    int itrack_ev = beg + itrack_plex;

    // TODO:: make sure that the width of the FitterCU is a multiple of
    //        Config::maxCandsPerSeed;
    int iseed_ev = itrack_ev / Config::maxCandsPerSeed;
    int icand_ev = itrack_ev % Config::maxCandsPerSeed;
    // | o : o : x : x : x |
    //  iseed
    //  <----> m_tracks_per_seed[iseed]
    //  <------------------> maxCandsPerSeed
    Valid(itrack_plex, 0, 0) = icand_ev < m_tracks_per_seed[iseed_ev]
                             && m_tracks_per_seed[iseed_ev] != 0;
    if (!Valid(itrack_plex, 0, 0)) {
      return;
    }

    Track &trk = tracks[beg];
    const char *varr       = (char*) &trk;
    int   off_error = (char*) trk.errArrayCU() - varr;
    int   off_param = (char*) trk.posArrayCU() - varr;

    int i= itrack_plex + beg;  // TODO: i == itrack_ev
    const Track &trk_i = tracks[i];
    int idx = (char*) &trk_i - varr;

    Label(itrack_plex, 0, 0) = tracks[i].label();
    SeedIdx(itrack_plex, 0, 0) = iseed_ev; 
    CandIdx(itrack_plex, 0, 0) = icand_ev;

    SlurpInIdx_fn(Err_iP, varr + off_error, idx, N);
    SlurpInIdx_fn(Par_iP, varr + off_param, idx, N);

    Chg(itrack_plex, 0, 0) = tracks[i].charge();
    Chi2(itrack_plex, 0, 0) = tracks[i].chi2();
    // Note Config::nLayers -- not suitable for building
    for (int hi = 0; hi < Nhits; ++hi) {
      HitsIdx[hi][itrack_plex] = tracks[i].getHitIdx(hi); 

      int hit_idx = HitsIdx[hi][itrack_plex];
    }
  }
}

__global__ 
void InputTracksAndHitIdxComb_kernel(Track *tracks, int *m_tracks_per_seed,
                                     GPlexLS Err_iP, GPlexLV Par_iP,
                                     GPlexQI Chg, GPlexQF Chi2,
                                     GPlexQI Label, GPlexQI *HitsIdx,
                                     GPlexQI SeedIdx, GPlexQI CandIdx,
                                     GPlexQB Valid, const int Nhits,
                                     const int beg, const int end, 
                                     const int N)
{
  int itrack = threadIdx.x + blockDim.x*blockIdx.x;
  InputTracksAndHitIdxComb_fn(tracks, m_tracks_per_seed, 
                              Err_iP, Par_iP,
                              Chg, Chi2, Label, HitsIdx, 
                              SeedIdx, CandIdx, Valid, Nhits ,
                              beg, end, itrack, N);
}

void InputTracksAndHitIdxComb_wrapper(const hipStream_t &stream, 
                                      const EtaBinOfCombCandidatesCU &etaBin,
                                     GPlexLS &Err_iP, GPlexLV &Par_iP,
                                     GPlexQI &Chg, GPlexQF &Chi2, 
                                     GPlexQI &Label, GPlexQI *HitsIdx,
                                     GPlexQI &SeedIdx, GPlexQI &CandIdx,
                                     GPlexQB &Valid, const int Nhits,
                                     const int beg, const int end,
                                     const bool inputProp, int N) {
  int gridx = std::min((N-1)/BLOCK_SIZE_X + 1,
                       max_blocks_x);
  dim3 grid(gridx, 1, 1);
  dim3 block(BLOCK_SIZE_X, 1, 1);

  InputTracksAndHitIdxComb_kernel<<< grid, block, 0, stream >>>
    (etaBin.m_candidates.data(), etaBin.m_ntracks_per_seed.data(),
     Err_iP, Par_iP, 
     Chg, Chi2, Label, HitsIdx,
     SeedIdx, CandIdx, Valid, Nhits,
     beg, end, N);
}
